#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/**
 * It computes the error of quaternions of the two postures, equivalent to TransformArchive::Error_q
 *
 */
#include "pch.h"
#include <algorithm>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "ik_logger.h"

// ComputeHErr_GPU<<<minGridSize, blockSize>>>(theta_q_dev, size_theta,
// 										 	err_out_dev, n_errs,
// 										 	n_joints, n_threads)

__global__ void
ComputeHErr_GPU(const Real4* theta_q, int n_theta, Real* err_out, int64_t n_err, int n_joints, int n_threads)
{
	for (int64_t i_err = blockDim.x * blockIdx.x + threadIdx.x
		; i_err < n_err
		; i_err += n_threads)
	{
		int64_t i_err2 = (i_err << 1);
		int i_theta1 = (int)floor(sqrtf(i_err2));
		int i_theta0 = i_theta1 + 1;		
		int i_theta = 0;
		
		if (!(i_err2 < i_theta1 * (i_theta1 + 1)))
			i_theta = i_theta0;
		else
		{
			i_theta = i_theta1;
			// IKAssert(i_theta * (i_theta - 1) <= i_err2);
		}	

		int j_theta2 = (int)(i_err2 - (int64_t)i_theta*(int64_t)(i_theta - 1));
		// IKAssert(0 == (j_theta2&0x01));
		int j_theta = (j_theta2 >> 1);
		// IKAssert(i_theta < n_theta); 
		// IKAssert(j_theta < n_theta);
		// IKAssert(j_theta < i_theta);
		// if (!(j_theta < i_theta))
		// {
		// 	LOGIKVarErr(LogInfoInt, i_theta);
		// 	LOGIKVarErr(LogInfoInt, j_theta);
		// }
		Real sigma_i_joint = 0;
		int i_theta_q_base = i_theta * n_joints;
		int j_theta_q_base = j_theta * n_joints;
		for (int i_joint = 0; i_joint < n_joints; i_joint ++)
		{
			auto q_i = theta_q[i_theta_q_base + i_joint];
			auto q_j = theta_q[j_theta_q_base + i_joint];
			auto err_k_ij = fabs( q_i.w * q_j.w
								+ q_i.x * q_j.x
								+ q_i.y * q_j.y
								+ q_i.z * q_j.z);
			sigma_i_joint += min((Real)1.0, err_k_ij);
		}
		Real err_i = (Real)n_joints - sigma_i_joint;
		err_out[i_err] = err_i;
	}
}


// ComputeXErr_GPU<<<minGridSize, blockSize>>>(theta0_q_dev, n_theta0,
// 												theta1_q_dev, n_theta1,
// 											 	err_out_dev, n_err,
// 											 	n_joints);

__global__ void
ComputeXErr_GPU(const Real4* theta0_q, int n_theta0, const Real4* theta1_q, int n_theta1, Real* err_out, int64_t n_err, int n_joints, int n_threads)
{
	for (int64_t i_err = blockDim.x * blockIdx.x + threadIdx.x
		; i_err < n_err
		; i_err += n_threads)
	{
		int i_theta0 = i_err / n_theta1;
		int i_theta1 = i_err % n_theta1;
		Real sigma_i_joint = 0;
		int i_theta0_q_base = i_theta0 * n_joints;
		int i_theta1_q_base = i_theta1 * n_joints;
		for (int i_joint = 0; i_joint < n_joints; i_joint ++)
		{
			auto q_0_i = theta0_q[i_theta0_q_base + i_joint];
			auto q_1_i = theta1_q[i_theta1_q_base + i_joint];
			auto err_k_ij = fabs( q_0_i.w * q_1_i.w
								+ q_0_i.x * q_1_i.x
								+ q_0_i.y * q_1_i.y
								+ q_0_i.z * q_1_i.z);
			sigma_i_joint += min((Real)1.0, err_k_ij);
		}
		Real err_i = (Real)n_joints - sigma_i_joint;
		err_out[i_err] = err_i;
	}
}

void ComputeXErr(const Real4* theta0_q, int n_theta0, const Real4* theta1_q, int n_theta1, Real* err_out, int64_t n_errs, int n_joints)
{
#if defined _GPU_PARALLEL

	#define RETURN_IF_F(condition)\
		if (condition)\
		{\
			const char* errInfo = hipGetErrorString(err);\
			LOGIKVarErr(LogInfoCharPtr, errInfo);\
			if (theta0_q_dev)\
				hipFree(theta0_q_dev);\
			if (theta1_q_dev)\
				hipFree(theta1_q_dev);\
			if (err_out_dev)\
				hipFree(err_out_dev);\
			return;\
		}


	size_t size_theta0 = n_theta0 * n_joints * sizeof(Real4);
	size_t size_theta1 = n_theta1 * n_joints * sizeof(Real4);
	size_t size_err = n_errs * sizeof(Real);

	Real4* theta0_q_dev = NULL;
	Real4* theta1_q_dev = NULL;
	Real* err_out_dev = NULL;
	hipError_t err = hipSuccess;

	RETURN_IF_F(hipSuccess != (err = hipMalloc((void **)&theta0_q_dev, size_theta0))
	 			|| hipSuccess != (err = hipMalloc((void **)&theta1_q_dev, size_theta1))
	 			|| hipSuccess != (err = hipMalloc((void **)&err_out_dev, size_err)));
	
	RETURN_IF_F(hipSuccess != (err = hipMemcpy(theta0_q_dev, theta0_q, size_theta0, hipMemcpyHostToDevice))
	 			|| hipSuccess != (err = hipMemcpy(theta1_q_dev, theta1_q, size_theta1, hipMemcpyHostToDevice)));
		

	int minGridSize = 0;
	int blockSize = 0;
	RETURN_IF_F(hipSuccess != (err = hipOccupancyMaxPotentialBlockSize(&minGridSize
																		, &blockSize
																		, (void*)ComputeXErr_GPU)));

	int n_threads = minGridSize * blockSize;
	// LOGIKVarErr(LogInfoInt, minGridSize);
	// LOGIKVarErr(LogInfoInt, blockSize);

	ComputeXErr_GPU<<<minGridSize, blockSize>>>(theta0_q_dev, n_theta0,
												theta1_q_dev, n_theta1,
											 	err_out_dev, n_errs,
											 	n_joints, n_threads);
	
	RETURN_IF_F(hipSuccess != (err = hipGetLastError()));
	
	RETURN_IF_F(hipSuccess != (err = hipMemcpy(err_out, err_out_dev, size_err, hipMemcpyDeviceToHost)));

	if (theta0_q_dev)
		hipFree(theta0_q_dev);
	if (theta1_q_dev)
		hipFree(theta1_q_dev);
	if (err_out_dev)
		hipFree(err_out_dev);

	#undef RETURN_IF_F

#else
	for (int i_err = 0; i_err < n_errs; i_err ++)
	{
		int i_theta0 = i_err / n_theta1;
		int i_theta1 = i_err % n_theta1;
		Real sigma_i_joint = 0;
		int i_theta0_q_base = i_theta0 * n_joints;
		int i_theta1_q_base = i_theta1 * n_joints;
		for (int i_joint = 0; i_joint < n_joints; i_joint ++)
		{
			auto q_0_i = theta0_q[i_theta0_q_base + i_joint];
			auto q_1_i = theta1_q[i_theta1_q_base + i_joint];
			auto err_k_ij = fabs( q_0_i.w * q_1_i.w
								+ q_0_i.x * q_1_i.x
								+ q_0_i.y * q_1_i.y
								+ q_0_i.z * q_1_i.z);
			sigma_i_joint += std::min((Real)1.0, err_k_ij);
		}
		Real err_i = (Real)n_joints - sigma_i_joint;
		err_out[i_err] = err_i;
	}
#endif
}


void ComputeHErr(const Real4* theta_q, int n_theta, Real* err_out, int64_t n_errs, int n_joints)
{
#if defined _GPU_PARALLEL

	#define RETURN_IF_F(condition)\
		if (condition)\
		{\
			const char* errInfo = hipGetErrorString(err);\
			LOGIKVarErr(LogInfoCharPtr, errInfo);\
			if (theta_q_dev)\
				hipFree(theta_q_dev);\
			if (err_out_dev)\
				hipFree(err_out_dev);\
			return;\
		}

	size_t size_theta = n_theta * n_joints * sizeof(Real4);
	size_t size_err = n_errs * sizeof(Real);

	Real4* theta_q_dev = NULL;
	Real* err_out_dev = NULL;
	hipError_t err = hipSuccess;

	RETURN_IF_F(hipSuccess != (err = hipMalloc((void **)&theta_q_dev, size_theta))
	 			|| hipSuccess != (err = hipMalloc((void **)&err_out_dev, size_err)));
	
	RETURN_IF_F(hipSuccess != (err = hipMemcpy(theta_q_dev, theta_q, size_theta, hipMemcpyHostToDevice)));
	 			
	int minGridSize = 0;
	int blockSize = 0;
	RETURN_IF_F(hipSuccess != (err = hipOccupancyMaxPotentialBlockSize(&minGridSize
																		, &blockSize
																		, (void*)ComputeHErr_GPU)));

	int n_threads = minGridSize * blockSize;
	// LOGIKVarErr(LogInfoInt, minGridSize);
	// LOGIKVarErr(LogInfoInt, blockSize);

	ComputeHErr_GPU<<<minGridSize, blockSize>>>(theta_q_dev, n_theta,
											 	err_out_dev, n_errs,
											 	n_joints, n_threads);
	
	RETURN_IF_F(hipSuccess != (err = hipGetLastError()));
	
	RETURN_IF_F(hipSuccess != (err = hipMemcpy(err_out, err_out_dev, size_err, hipMemcpyDeviceToHost)));

	if (theta_q_dev)
		hipFree(theta_q_dev);
	if (err_out_dev)
		hipFree(err_out_dev);

	#undef RETURN_IF_F
#else
	for (int64_t i_err = 0; i_err < n_errs; i_err ++)
	{
		int64_t i_err2 = (i_err << 1);
		int i_theta1 = (int)std::floor(std::sqrt(i_err2));
		int i_theta0 = i_theta1 + 1;		
		int i_theta = 0;
		
		if (!(i_err2 < i_theta1 * (i_theta1 + 1)))
			i_theta = i_theta0;
		else
		{
			i_theta = i_theta1;
			IKAssert(i_theta * (i_theta - 1) <= i_err2);
		}	

		int j_theta2 = (int)(i_err2 - (int64_t)i_theta*(int64_t)(i_theta - 1));
		IKAssert(0 == (j_theta2&0x01));
		int j_theta = (j_theta2 >> 1);
		IKAssert(i_theta < n_theta); 
		IKAssert(j_theta < n_theta);
		IKAssert(j_theta < i_theta);
		if (!(j_theta < i_theta))
		{
			LOGIKVarErr(LogInfoInt, i_theta);
			LOGIKVarErr(LogInfoInt, j_theta);
		}
		Real sigma_i_joint = 0;
		int i_theta_q_base = i_theta * n_joints;
		int j_theta_q_base = j_theta * n_joints;
		for (int i_joint = 0; i_joint < n_joints; i_joint ++)
		{
			auto q_i = theta_q[i_theta_q_base + i_joint];
			auto q_j = theta_q[j_theta_q_base + i_joint];
			auto err_k_ij = fabs( q_i.w * q_j.w
								+ q_i.x * q_j.x
								+ q_i.y * q_j.y
								+ q_i.z * q_j.z);
			sigma_i_joint += std::min((Real)1.0, err_k_ij);
		}
		Real err_i = (Real)n_joints - sigma_i_joint;
		err_out[i_err] = err_i;
	}
#endif
}
