#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/**
 * Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 2
 * of the programming guide with some additions like error checking.
 */
#include "pch.h"
#include <algorithm>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "ik_logger.h"

// ComputeErr_GPU<<<minGridSize, blockSize>>>(theta0_q_dev, n_theta0,
// 												theta1_q_dev, n_theta1,
// 											 	err_out_dev, n_err,
// 											 	n_joints);

__global__ void
ComputeErr_GPU(const Real4* theta0_q, int n_theta0, const Real4* theta1_q, int n_theta1, Real* err_out, int64_t n_err, int n_joints, int n_threads)
{
	for (int i_err = blockDim.x * blockIdx.x + threadIdx.x
		; i_err < n_err
		; i_err += n_threads)
	{
		int i_theta0 = i_err / n_theta1;
		int i_theta1 = i_err % n_theta1;
		Real sigma_k = 0;
		for (int i_joint = 0; i_joint < n_joints; i_joint ++)
		{
			auto q_0_i = theta0_q[i_theta0 * n_joints + i_joint];
			auto q_1_i = theta1_q[i_theta1 * n_joints + i_joint];
			auto err_k_ij = fabs( q_0_i.w * q_1_i.w
								+ q_0_i.x * q_1_i.x
								+ q_0_i.y * q_1_i.y
								+ q_0_i.z * q_1_i.z);
			sigma_k += min((Real)1.0, err_k_ij);
		}
		Real err_i = (Real)n_joints - sigma_k;
		err_out[i_err] = err_i;
	}
}

void ComputeErr(const Real4* theta0_q, int n_theta0, const Real4* theta1_q, int n_theta1, Real* err_out, int64_t n_err, int n_joints)
{
#if defined _GPU_PARALLEL
	size_t size_theta0 = n_theta0 * n_joints * sizeof(Real4);
	size_t size_theta1 = n_theta1 * n_joints * sizeof(Real4);
	size_t size_err = n_err * sizeof(Real);

	Real4* theta0_q_dev = NULL;
	Real4* theta1_q_dev = NULL;
	Real* err_out_dev = NULL;
	hipError_t err = hipSuccess;
	if (hipSuccess != (err = hipMalloc((void **)&theta0_q_dev, size_theta0))
	 || hipSuccess != (err = hipMalloc((void **)&theta1_q_dev, size_theta1))
	 || hipSuccess != (err = hipMalloc((void **)&err_out_dev, size_err)))
	{
		const char* errInfo = hipGetErrorString(err);
		LOGIKVarErr(LogInfoCharPtr, errInfo);
		if (theta0_q_dev)
			hipFree(theta0_q_dev);
		if (theta1_q_dev)
			hipFree(theta1_q_dev);
		if (err_out_dev)
			hipFree(err_out_dev);
		return;
	}
	
	if (hipSuccess != (err = hipMemcpy(theta0_q_dev, theta0_q, size_theta0, hipMemcpyHostToDevice))
	 || hipSuccess != (err = hipMemcpy(theta1_q_dev, theta1_q, size_theta1, hipMemcpyHostToDevice)))
	{
		const char* errInfo = hipGetErrorString(err);
		LOGIKVarErr(LogInfoCharPtr, errInfo);
		if (theta0_q_dev)
			hipFree(theta0_q_dev);
		if (theta1_q_dev)
			hipFree(theta1_q_dev);
		if (err_out_dev)
			hipFree(err_out_dev);		
		return;
	}

	int minGridSize = 0;
	int blockSize = 0;
	if (hipSuccess != (err = hipOccupancyMaxPotentialBlockSize(&minGridSize
																, &blockSize
																, (void*)ComputeErr_GPU)))
	{
		const char* errInfo = hipGetErrorString(err);
		LOGIKVarErr(LogInfoCharPtr, errInfo);
		if (theta0_q_dev)
			hipFree(theta0_q_dev);
		if (theta1_q_dev)
			hipFree(theta1_q_dev);
		if (err_out_dev)
			hipFree(err_out_dev);		
		return;
	}

	int n_threads = minGridSize * blockSize;
	// LOGIKVarErr(LogInfoInt, minGridSize);
	// LOGIKVarErr(LogInfoInt, blockSize);

	ComputeErr_GPU<<<minGridSize, blockSize>>>(theta0_q_dev, n_theta0,
												theta1_q_dev, n_theta1,
											 	err_out_dev, n_err,
											 	n_joints, n_threads);
	
	if (hipSuccess != (err = hipGetLastError()))
	{
		const char* errInfo = hipGetErrorString(err);
		LOGIKVarErr(LogInfoCharPtr, errInfo);
		if (theta0_q_dev)
			hipFree(theta0_q_dev);
		if (theta1_q_dev)
			hipFree(theta1_q_dev);
		if (err_out_dev)
			hipFree(err_out_dev);
		return;		
	}
	
	if (hipSuccess != (err = hipMemcpy(err_out, err_out_dev, size_err, hipMemcpyDeviceToHost)))
	{
		const char* errInfo = hipGetErrorString(err);
		LOGIKVarErr(LogInfoCharPtr, errInfo);
		if (theta0_q_dev)
			hipFree(theta0_q_dev);
		if (theta1_q_dev)
			hipFree(theta1_q_dev);
		if (err_out_dev)
			hipFree(err_out_dev);
		return;
	}

	if (theta0_q_dev)
		hipFree(theta0_q_dev);
	if (theta1_q_dev)
		hipFree(theta1_q_dev);
	if (err_out_dev)
		hipFree(err_out_dev);

#else
	for (int i_err = 0; i_err < n_err; i_err ++)
	{
		int i_theta0 = i_err / n_theta1;
		int i_theta1 = i_err % n_theta1;
		Real sigma_k = 0;
		for (int i_joint = 0; i_joint < n_joints; i_joint ++)
		{
			auto q_0_i = theta0_q[i_theta0 * n_joints + i_joint];
			auto q_1_i = theta1_q[i_theta1 * n_joints + i_joint];
			auto err_k_ij = fabs( q_0_i.w * q_1_i.w
								+ q_0_i.x * q_1_i.x
								+ q_0_i.y * q_1_i.y
								+ q_0_i.z * q_1_i.z);
			sigma_k += std::min((Real)1.0, err_k_ij);
		}
		Real err_i = (Real)n_joints - sigma_k;
		err_out[i_err] = err_i;
	}
#endif
}





